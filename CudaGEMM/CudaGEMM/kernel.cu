﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#define M 512 
#define K 512
#define N 512

void initial(float* array, int size)
{
	for (int i = 0; i < size; i++)
	{
		array[i] = (float)(rand() % 10 + 1);
	}
}

__global__ void mulitplicateMatrix(float* array_A, float* array_B, float *array_C, int M_p, int K_p, int N_p)
{
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iy = blockDim.y * blockIdx.y + threadIdx.y;
	if (ix < N_p && iy < N_p)
	{
		float sum = 0;
		for (int k = 0; k < K_p; k++)
		{
			sum = array_A[iy * K_p + k] * array_B[ix * N_p + k];
		}
		array_C[iy* M_p + ix] = sum;
	}

}

int main()
{
	int Axy = M * K;
	int Bxy = K * N;
	int Cxy = M * N;
	
	float* h_A, * h_B,*h_C, * hostRef, * deviceRef;
	h_A = (float*)malloc(Axy * sizeof(float));
	h_B = (float*)malloc(Bxy * sizeof(float));
	h_C = (float*)malloc(Cxy * sizeof(float));
	initial(h_A, Axy);
	initial(h_B, Axy);

	float* d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, Axy * sizeof(float));
	hipMalloc((void**)&d_B, Bxy * sizeof(float));
	hipMalloc((void**)&d_C, Cxy * sizeof(float));

	hipMemcpy(d_A, h_A, Axy * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, Bxy * sizeof(float), hipMemcpyHostToDevice);

	dim3 block(2,2);
	dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);
	
	mulitplicateMatrix <<<grid, block >>> (d_A, d_B, d_C,M,K,N);
	hipMemcpy(h_C, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}